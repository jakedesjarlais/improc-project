#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/**
 * CUDA kernels for transforming color spaces from ARGB -> NV12
 *
 * NOTE: THESE KERNELS WERE PULLED FROM NVIDIA'S GITHUB
 * I did not write these
 *
 */


__forceinline__ __device__
float clamp(float x, float a, float b) {
  return max(a, min(b, x));
}

__forceinline__  __device__
float RGBA2Y(uchar4  argb)
{
  return clamp((0.257*argb.x) + (0.504*argb.y) + (0.098*argb.z) + 16, 0, 255);
}

__global__
static void CudaProcessY(int w, int h, uchar4 * pARGBImage, unsigned char * pNV12ImageY) {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y; 

    if (i < w && j < h) {
        uchar4 argb=pARGBImage[w*j +k i];
        pNV12ImageY[w*j + i]= RGBA2Y(argb);
    }
}

__forceinline__  __device__
float RGBA2U(uchar4  argb) {
  return clamp(-(0.148*argb.x) - (0.291*argb.y) + (0.439*argb.z) + 128.0, 0, 255);
}

__forceinline__  __device__
float RGBA2V(uchar4  argb) {
  return clamp((0.439*argb.x) - (0.368*argb.y) - (0.0701*argb.z) + 128.0, 0, 255);
}

__global__
static void CudaProcessUV(int w,
                          int h,
                          uchar4* pARGBImage,
                          unsigned char* pNV12ImageUV) {

    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y; 
    unsigned int fi = i*2;//full size image i
    unsigned int fj = j*2;//full size image j
    unsigned int fw = w*2;//full size image w
    unsigned int fh = h*2;//full size image h
    unsigned int u_idx = i*2 + 1 + j*w*2;
    unsigned int v_idx = i*2 + j*w*2;

    if(fi<fw-1 && fj<fh-1) {
        uchar4 argb1 = pARGBImage[fj*fw + fi];
        uchar4 argb2 = pARGBImage[fj*fw + fi + 1];
        uchar4 argb3 = pARGBImage[(fj + 1)*fw + fi];
        uchar4 argb4 = pARGBImage[(fj + 1)*fw + fi+1];

        float U  = RGBA2U(argb1);
        float U2 = RGBA2U(argb2);
        float U3 = RGBA2U(argb3);
        float U4 = RGBA2U(argb4);

        float V =  RGBA2V(argb1);
        float V2 = RGBA2V(argb2);
        float V3 = RGBA2V(argb3);
        float V4 = RGBA2V(argb4);

        pNV12ImageUV[u_idx] = (U+U2+U3+U4)/4.0;
        pNV12ImageUV[v_idx] = (V+V2+V3+V4)/4.0;
    }
}


// Need extern C here to allow access from main program
// CUDA is weird
extern "C" {

  hipError_t launch_CudaARGB2NV12Process(int w,
                                        int h,
                                        hipDeviceptr_t pARGBImage,
                                        hipDeviceptr_t pNV12Image) {
    {
      dim3 dimBlock(16, 16, 1);
      dim3 dimGrid(((w) + dimBlock.x - 1)/dimBlock.x, ((h) + dimBlock.y - 1)/dimBlock.y, 1);
      CudaProcessY<<<dimGrid, dimBlock>>>(w, h, (uchar4*)pARGBImage, (unsigned char*)pNV12Image);   
    }
    {
      dim3 dimBlock(16, 16, 1);
      dim3 dimGrid(((w/2) + dimBlock.x - 1)/dimBlock.x, ((h/2) + dimBlock.y - 1)/dimBlock.y, 1);
      CudaProcessUV<<<dimGrid, dimBlock>>>(w/2, h/2, (uchar4*)pARGBImage, ((unsigned char*)pNV12Image) + w*h);        
    }
    hipError_t err = hipGetLastError();                                
    return err;
  }
}
